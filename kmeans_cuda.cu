// File: kmeans_cuda.cu
// Descrição: K-means adaptado para execução em GPU via CUDA com saída de clusters

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

// ─────────── CONFIGURAÇÃO ───────────
#define DATA_FILE       "covtype.csv"    // Arquivo CSV de entrada
#define DEFAULT_K       10               // Número de clusters
#define DEFAULT_MAX_IT 150              // Máximo de iterações
#define SKIP_HEADER     true             // Pular primeira linha (header)
// ─────────────────────────────────────

// Função para carregar CSV no host (retorna vector[N][D])
vector<vector<double>> load_csv(const string& filename) {
    ifstream in(filename);
    if (!in) {
        cerr << "Erro ao abrir arquivo: " << filename << "\n";
        exit(1);
    }
    vector<vector<double>> data;
    string line;
#if SKIP_HEADER
    getline(in, line);  // descarta header
#endif
    while (getline(in, line)) {
        while (!line.empty() && (line.back()=='\r' || line.back()=='\n' || line.back()==','))
            line.pop_back();
        if (line.empty()) continue;
        stringstream ss(line);
        vector<double> row;
        string cell;
        while (getline(ss, cell, ',')) {
            try { row.push_back(stod(cell)); } catch (...) {}
        }
        if (!row.empty()) row.pop_back();
        if (!row.empty()) data.emplace_back(move(row));
    }
    return data;
}

// Kernel CUDA: atribui cada ponto ao cluster mais próximo
__global__ void assign_labels(const double* data,
                              const double* centroids,
                              int* labels,
                              int N, int D, int K) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    const double* p = data + idx * D;
    double best_dist = 1e300;
    int best_k = 0;
    for (int k = 0; k < K; ++k) {
        const double* c = centroids + k * D;
        double dist = 0;
        #pragma unroll
        for (int d = 0; d < D; ++d) {
            double diff = p[d] - c[d];
            dist += diff * diff;
        }
        if (dist < best_dist) { best_dist = dist; best_k = k; }
    }
    labels[idx] = best_k;
}

int main(int argc, char* argv[]) {
    int K = (argc >= 2 ? stoi(argv[1]) : DEFAULT_K);
    int max_iter = (argc >= 3 ? stoi(argv[2]) : DEFAULT_MAX_IT);

    // Carrega dados no host
    auto host_data = load_csv(DATA_FILE);
    int N = host_data.size();
    int D = host_data[0].size();
    cout << "→ Carreguei " << N << " amostras (dim=" << D << ")\n";

    // Flatten dados em vetor contínuo
    vector<double> flat_data(N * D);
    for (int i = 0; i < N; ++i) {
        memcpy(flat_data.data() + i * D, host_data[i].data(), D * sizeof(double));
    }

    // Host: centroids e labels
    vector<double> h_centroids(K * D);
    vector<int> h_labels(N, -1);

    // Inicialização randômica dos centróides
    mt19937_64 rng(1234);
    uniform_int_distribution<int> pick(0, N - 1);
    unordered_set<int> used;
    for (int k = 0; k < K; ) {
        int idx = pick(rng);
        if (used.insert(idx).second) {
            copy_n(flat_data.data() + idx * D, D, h_centroids.begin() + k * D);
            ++k;
        }
    }

    // Alocação de memória na GPU
    double *d_data, *d_centroids;
    int *d_labels;
    hipMalloc(&d_data, N * D * sizeof(double));
    hipMalloc(&d_centroids, K * D * sizeof(double));
    hipMalloc(&d_labels, N * sizeof(int));

    // Cópia inicial de dados para GPU
    hipMemcpy(d_data, flat_data.data(), N * D * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Loop principal K-means
    for (int it = 0; it < max_iter; ++it) {
        hipMemcpy(d_centroids, h_centroids.data(), K * D * sizeof(double), hipMemcpyHostToDevice);
        assign_labels<<<blocks, threadsPerBlock>>>(d_data, d_centroids, d_labels, N, D, K);
        hipDeviceSynchronize();
        hipMemcpy(h_labels.data(), d_labels, N * sizeof(int), hipMemcpyDeviceToHost);

        // Recalcula centróides no host
        vector<vector<double>> sum(K, vector<double>(D, 0.0));
        vector<int> count(K, 0);
        bool changed = false;
        for (int i = 0; i < N; ++i) {
            int lbl = h_labels[i];
            ++count[lbl];
            for (int d = 0; d < D; ++d) {
                sum[lbl][d] += flat_data[i * D + d];
            }
        }
        for (int k = 0; k < K; ++k) {
            if (count[k] > 0) {
                for (int d = 0; d < D; ++d) {
                    double new_val = sum[k][d] / count[k];
                    if (fabs(new_val - h_centroids[k * D + d]) > 1e-6) changed = true;
                    h_centroids[k * D + d] = new_val;
                }
            }
        }
        if (!changed) {
            cout << "Convergiu em " << it << " iterações.\n";
            break;
        }
    }

    // Saída dos centróides
    cout << fixed << setprecision(4);
    for (int k = 0; k < K; ++k) {
        cout << "Centróide " << k << ": ";
        for (int d = 0; d < D; ++d) cout << h_centroids[k * D + d] << " ";
        cout << "\n";
    }

    // Impressão dos tamanhos de cada cluster
    vector<int> final_count(K, 0);
    for (int lbl : h_labels) ++final_count[lbl];
    for (int k = 0; k < K; ++k) {
        cout << "Cluster " << k << " tem " << final_count[k] << " pontos\n";
    }

    // Liberação de memória GPU
    hipFree(d_data);
    hipFree(d_centroids);
    hipFree(d_labels);
    return 0;
}
